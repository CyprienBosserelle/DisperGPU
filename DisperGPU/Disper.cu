#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//DisperGPU   v0.0                                                                 //
//Copyright (C) 2015 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////


#include "Header.cuh"

#include "Disper_kernel.cu"

FILE * logfile;

float4 * partpos,*partpos_g; //Particule position x,y,z,t


float *Uo, *Un; //U velocity, Step 0 and step n
float *Vo, *Vn; //V velocity, Step 0 and step n
float *Ux, *Vx, *hhx; // U and V velocity at the dispersal step
float *hho, *hhn; // Water depth, Step 0 and step n
float *Uo_g, *Un_g, *Ux_g; //Same on GPU plus at t particle step
float *Vo_g, *Vn_g, *Vx_g; // Same on GPU plus at t particle step
float *hho_g, *hhn_g, *hhx_g;// Same on GPU plus at t particle step

float *Nincel, *cNincel, *cTincel; // Number of particle in cell, Cumulative Nincel, Cumulative time in cell CPU
float *Nincel_g, *cNincel_g, *cTincel_g; // Number of particle in cell, Cumulative Nincel, Cumulative time in cell on GPU

float *distX, *distY; // Distance calculated between cells
float *xcoord, *ycoord; // REal world coordinates


float * d_Rand; //GPU random number array
hiprandGenerator_t gen; // Random number generator using hiprand

hipError_t CUDerr; // Catching CUDA errors 

hipArray* Ux_gp; // Cuda array to pre-store HD vel data before converting to textures
hipArray* Vx_gp; // Cuda array to pre-store HD vel data before converting to textures
hipArray* hhx_gp; // Cuda array to pre-store HD depth data before converting to textures
hipArray* distX_gp; // Cuda array to pre-store HD distance before converting to textures
hipArray* distY_gp; // Cuda array to pre-store HD distance before converting to textures

hipArray* xcoord_gp; // Cuda array to pre-store HD coordinates before converting to textures
hipArray* ycoord_gp; // Cuda array to pre-store HD coordinates before converting to textures

// Below create channels between cuda arrays (see above) and textures
hipChannelFormatDesc channelDescU = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescV = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDeschh = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescdX = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescdY = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescxcoord = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescycoord = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);


void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}
void GPUstep()
{
	dim3 blockDimHD(16, 16, 1);
	//dim3 gridDimHD(ceil(max(netau,netav)*max(nxiu,nxiv) / (float)blockDimHD.x), 1, 1);
	dim3 gridDimHD((int)ceil((float)nx / (float)blockDimHD.x), (int)ceil((float)ny / (float)blockDimHD.y), 1);

	if (totaltime >= hddt*(hdstep - hdstart + 1))//+1 because we only read the next step when time exeed the previous next step
	{
		//Read next step
		printf("Reading Next step\n");
		hdstep++;

		int steptoread = hdstep;

		if (backswitch>0)
		{
			steptoread = hdend - hdstep;
		}

		NextHDstep<<<gridDimHD, blockDimHD, 0 >>>(nx, ny, Uo_g, Un_g);
		CUDA_CHECK(hipDeviceSynchronize());

		NextHDstep<<<gridDimHD, blockDimHD, 0 >>>(nx, ny, Vo_g, Vn_g);
		CUDA_CHECK(hipDeviceSynchronize());

		//NextHDstep<<<gridDimHD, blockDimHD, 0 >>>(nx, ny, hho_g, hhn_g);
		//CUDA_CHECK(hipDeviceSynchronize());

		
		//readHDstepHYCOM(ncfile, Uvarname, Vvarname, nx, ny, steptoread, lev, Un, Vn, hhn);
		readHDstep(ncfile, Uvarname, Vvarname, hhvarname, nx, ny, steptoread, lev, Un, Vn, hhn);

		CUDA_CHECK(hipMemcpy(Un_g, Un, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(Vn_g, Vn, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		//CUDA_CHECK(hipMemcpy(hhn_g, hhn, nx*ny*sizeof(float), hipMemcpyHostToDevice));

	}
	//printf("Run GPU step\n");

	//printf("Nincel Reset\n");
	ResetNincel<<<gridDimHD, blockDimHD, 0 >>>(nx, ny, Nincel_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//printf("interp HD\n");
	//int interpstep = hdstep - hdstart + 1;
	//InterpstepCPU(nx, ny, backswitch, hdstep, totaltime, hddt, Ux, Uo, Un);
	HD_interp <<< gridDimHD, blockDimHD, 0 >>>(nx, ny, backswitch, hdstep - hdstart, totaltime, hddt, Uo_g, Un_g, Ux_g);
	CUDA_CHECK(hipDeviceSynchronize());

	HD_interp << <gridDimHD, blockDimHD, 0 >>>(nx, ny, backswitch, hdstep - hdstart, totaltime, hddt, Vo_g, Vn_g, Vx_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//HD_interp <<<gridDimHD, blockDimHD, 0 >>>(nx, ny, backswitch, hdstep, totaltime, hddt, hho_g, hhn_g, hhx_g);
	//CUDA_CHECK(hipDeviceSynchronize());


	//printf("Mem copy to array\n");
	CUDA_CHECK(hipMemcpyToArray(Ux_gp, 0, 0, Ux_g, nx*ny* sizeof(float), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpyToArray(Vx_gp, 0, 0, Vx_g, nx*ny* sizeof(float), hipMemcpyDeviceToDevice));
	//CUDA_CHECK(hipMemcpyToArray(hhx_gp, 0, 0, hhx_g, nx*ny* sizeof(float), hipMemcpyDeviceToDevice));
	//Generate some random numbers
	// Set seed 
	//hiprandSetPseudoRandomGeneratorSeed(gen, SEED);
	// Generate n floats on device 

	//printf("Rnd gen\n");
	hiprandGenerateUniform(gen, d_Rand, np);

	//printf("Part position\n");
	//run the model
	//int nbblocks=npart/256;
	dim3 blockDim(256, 1, 1);
	dim3 gridDim(np / blockDim.x, 1, 1);

	//Calculate particle step

	updatepartpos <<<gridDim, blockDim, 0 >>>(np, dt, Eh, d_Rand, partpos_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//ij2lonlat <<<gridDim, blockDim, 0 >> >(np, xl_g, yl_g, xp_g, yp_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("Calc Nincel\n");
	CalcNincel <<<gridDim, blockDim, 0 >>>(np, nx, ny, partpos_g, Nincel_g, cNincel_g, cTincel_g);
	CUDA_CHECK(hipDeviceSynchronize());
}




void CPUstep()
{
	if (totaltime >= hddt*(hdstep - hdstart + 1))//+1 because we only read the next step when time exeed the previous next step
	{
		//Read next step

		hdstep++;

		int steptoread = hdstep;

		if (backswitch>0)
		{
			steptoread = hdend - hdstep;
		}

		NextstepCPU(nx,ny, Uo, Vo, hho, Un, Vn, hhn);
		//readHDstepHYCOM(ncfile, Uvarname, Vvarname, nx, ny, steptoread, lev, Un, Vn, hhn);
		readHDstep(ncfile, Uvarname, Vvarname, hhvarname, nx, ny, steptoread, lev, Un, Vn, hhn);


	}

	//Interpolate U vel
	InterpstepCPU( nx, ny, backswitch, hdstep, totaltime, hddt, Ux, Uo, Un);

	//Interpolate V vel
	InterpstepCPU(nx, ny, backswitch, hdstep, totaltime, hddt, Vx, Vo, Vn);

	//Interpolate Water depth
	InterpstepCPU(nx, ny, 1.0f, hdstep, totaltime, hddt, hhx, hho, hhn);

	// Reseed random number
	//not needed here?

	// Update particle position
	updatepartposCPU(nx, ny, np, dt, Eh, Ux, Vx, hhx, distX, distY, partpos);

	//update Nincel
	calcNincelCPU(np, nx, ny, partpos, Nincel, cNincel, cTincel);


}



int main()
{
	//Model starts Here//

	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	clock_t startcputime, endcputime;


	startcputime = clock();
	Param Dparam;
	Control Dcontrol;

	// Initialise totaltime
	Dcontrol.totaltime = 0.0;
	Dcontrol.nextouttime = 0.0;

	char logfilename[] = "DisperGPU.log";
	logfile = fopen(logfilename, "w");
	fprintf(logfile, "DisperGPU v0.0\n");


	//////////////////////////////////////////////////////
	/////             Read Operational file           /////
	//////////////////////////////////////////////////////
	fprintf(logfile, "Reading DisperGPU.dat...\t");
	printf( "Reading DisperGPU.dat\n");





	char opfile[] = "DisperGPU.dat";
	
	


	FILE * fop;
	fop = fopen(opfile, "r");

	if (fop == NULL)
	{
		fprintf(logfile, "Error opening DisperGPU.dat: %s\n", strerror(errno));
		perror("Error opening DisperGPU.dat: ");

		exit(-1);
	}
	

	
	fscanf(fop, "%*s %s\t%*s", &Dparam.ncfile); //HD file name should have U V velocity and depth
	fscanf(fop, "%s\t%*s", &Dparam.Uvarname);
	fscanf(fop, "%s\t%*s", &Dparam.Vvarname);
	fscanf(fop, "%s\t%*s", &Dparam.hhvarname);
	fscanf(fop, "%f\t%*s", &Dparam.hddt);
	fscanf(fop, "%d,%d\t%*s", &Dparam.hdstart, &Dparam.hdend);
	fscanf(fop, "%d\t%*s", &Dparam.lev);
	fscanf(fop, "%d\t%*s", &Dparam.geocoord);
	fscanf(fop, "%d\t%*s", &Dparam.backswitch);
	fscanf(fop, "%d\t%*s", &Dparam.partmode);
	fscanf(fop, "%u\t%*s", &Dparam.np);
	fscanf(fop, "%f\t%*s", &Dcontrol.dt);
	fscanf(fop, "%f\t%*s", &Dparam.Eh);
	fscanf(fop, "%f\t%*s", &Dparam.Ev);
	fscanf(fop, "%f\t%*s", &Dparam.minrwdepth);
	fscanf(fop, "%s\t%*s", &Dparam.seedfile);
	//fscanf(fop, "%d\t%*s", &GPUDEV);

	//fscanf(fop, "%d\t%*s", &outtype);
	fscanf(fop, "%f\t%*s", &Dcontrol.outtime);
	fscanf(fop, "%s\t%*s", &Dparam.ncoutfile);

	fclose(fop);

	fprintf(logfile, "Complete\n");
	fprintf(logfile, "Reading netCDF file : %s...\n", Dparam.ncfile);
	printf("Reading netCDF file: %s...\n", Dparam.ncfile);
	//readgridsizeHYCOM(ncfile, Uvarname, Vvarname, nt, nx, ny, xcoord, ycoord);
	readgridsize(Dparam.ncfile, Dparam.Uvarname, Dparam.Vvarname, Dparam.hhvarname, Dparam.nt, Dparam.nx, Dparam.ny, xcoord, ycoord);


	fprintf(logfile, "\t nx=%d\tny=%d\tnt=%d\n", Dparam.nx, Dparam.ny, Dparam.nt);
	printf("\t nx=%d\tny=%d\tnt=%d\n", Dparam.nx, Dparam.ny, Dparam.nt);
	fprintf(logfile, "...done\n");
	printf("...done\n");

	int nx = Dparam.nx;
	int ny = Dparam.ny;
	int nt = Dparam.nt;


	//set up CPU mem
	printf("Allocate CPU memory... ");
	//Vel ARRAYS
	Uo = (float *)malloc(nx*ny*sizeof(float));
	Un = (float *)malloc(nx*ny*sizeof(float));
	Vo = (float *)malloc(nx*ny*sizeof(float));
	Vn = (float *)malloc(nx*ny*sizeof(float));
	hho = (float *)malloc(nx*ny*sizeof(float));
	hhn = (float *)malloc(nx*ny*sizeof(float));

	Ux = (float *)malloc(nx*ny*sizeof(float));
	Vx = (float *)malloc(nx*ny*sizeof(float));
	hhx = (float *)malloc(nx*ny*sizeof(float));

	distX = (float *)malloc(nx*ny*sizeof(float));
	distY = (float *)malloc(nx*ny*sizeof(float));

	/* initialize random seed: */
	srand((unsigned int)time(NULL));


	//xcoord = (float *)malloc(nx*ny*sizeof(float));// Already allocated in readgridsize subroutine
	//ycoord = (float *)malloc(nx*ny*sizeof(float));

	//Nincel
	Nincel = (float *)malloc(nx*ny*sizeof(float));
	cNincel = (float *)malloc(nx*ny*sizeof(float));
	cTincel = (float *)malloc(nx*ny*sizeof(float));


	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			Nincel[i + j*nx] = 0.0f;
			cNincel[i + j*nx] = 0.0f;
			cTincel[i + j*nx] = 0.0f;
		}
	}
	printf("...done\n");

	printf("Calculate distance array... ");
	CalcDistXY(nx, ny, Dparam.geocoord, xcoord, ycoord, distX,distY);
	printf("...done\n");
	//Calculate first HD step
	//outstep=10;
	Dcontrol.stp = 0;//hdstart*hddt/dt;
	Dcontrol.hdstep = Dcontrol.hdstart;


	
	//printf("HD step:%d\n ",hdstep);
	if (Dcontrol.hdend == 0)
	{
		Dcontrol.hdend = Dparam.nt - 1;
	}

	if (Dcontrol.outtime == 0.0f)
	{
		Dcontrol.outtime = Dparam.hddt*Dcontrol.hdend;
	}
	Dcontrol.nextouttime = Dcontrol.outtime;

	int steptoread = Dcontrol.hdstep;

	if (Dparam.backswitch>0)
	{
		steptoread = Dcontrol.hdend - Dcontrol.hdstep;
	}

	
	//////////////////////////////
	//Read first step in Hd model
	///////////////////////////////

	//readHDstepHYCOM(ncfile, Uvarname, Vvarname, nx, ny, steptoread, lev, Uo, Vo, hho);
	
	
	//Also read next step?
	//readHDstepHYCOM(ncfile, Uvarname, Vvarname, nx, ny, steptoread+1, lev, Un, Vn, hhn);

	readHDstep(Dparam.ncfile, Dparam.Uvarname, Dparam.Vvarname, Dparam.hhvarname, Dparam.nx, Dparam.ny, steptoread, Dparam.lev, Uo, Vo, hho);

	//Also read next step?
	readHDstep(Dparam.ncfile, Dparam.Uvarname, Dparam.Vvarname, Dparam.hhvarname, Dparam.nx, Dparam.ny, steptoread + 1, Dparam.lev, Un, Vn, hhn);

	//Calculate best dt
	if (!(Dcontrol.dt > 0.0f))// if dt==0.0
	{
		Calcmaxstep(nx, ny, Dcontrol.dt, Dparam.hddt, Uo, Vo, Un, Vn, distX, distY);
	}
	Dcontrol.olddt = Dcontrol.dt;
	printf("Allocating CPU memory for particle position... ");
	//Initialise particles on CPU
	partpos = (float4 *)malloc(Dparam.np*sizeof(float4));
	d_Rand = (float *)malloc(Dparam.np*sizeof(float));

	//partpos[50] = make_float4(0.0f, 1.0f, 5.0f, 0.2);
	printf("...done.\n");
	//printf("partpos.x=%f", partpos[50].z);
	


	//printf("partpos.x=%f", partpos[50].x);
	//Find GPU
	int nDevices;

	hipGetDeviceCount(&nDevices);// Crash when using CUDA check?
	//GPUDEV = -1;
	if (nDevices > 0)
	{
		printf("(%i) Cuda device(s) found!\n",nDevices);
	}
	else
	{
		printf("No GPU found. Using CPU only\n");
		Dparam.GPUDEV = -1;
	}

	if (Dparam.GPUDEV > nDevices && Dparam.GPUDEV>0)
	{
		printf("Specified GPU Device not found, Using Device %i.\n",0);
		Dparam.GPUDEV = 0;
	}
	if (Dparam.GPUDEV >= 0)
	{
		printf("Allocating mem on GPU...");
		CUDA_CHECK(hipSetDevice(Dparam.GPUDEV)); //Add error handling
		//If GPU available then copy set up GPU mem
		

		CUDA_CHECK(hipMalloc((void **)&partpos_g, np*sizeof(float4)));

		CUDA_CHECK(hipMalloc((void **)&Uo_g, nx*ny* sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Un_g, nx*ny* sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Ux_g, nx*ny* sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&Vo_g, nx*ny* sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Vn_g, nx*ny* sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Vx_g, nx*ny* sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&Nincel_g, nx*ny* sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&cNincel_g, nx*ny* sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&cTincel_g, nx*ny* sizeof(float)));


		printf(" ...done\n");

		// Loading random number generator
		hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

		CUDA_CHECK(hipMalloc((void **)&d_Rand, np*sizeof(float)));

		printf("Transfert vectors to GPU memory... ");
		CUDA_CHECK(hipMemcpy(Uo_g, Uo, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(Un_g, Uo, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(Ux_g, Uo, nx*ny*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK(hipMemcpy(Vo_g, Vo, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(Vn_g, Vo, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(Vx_g, Vo, nx*ny*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK(hipMemcpy(Nincel_g, Nincel, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(cNincel_g, Nincel, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(cTincel_g, Nincel, nx*ny*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK(hipMemcpy(partpos_g, partpos, np*sizeof(float4), hipMemcpyHostToDevice));

		//CUDA_CHECK(hipMemcpy(partpos_g, partpos, np*sizeof(float4), hipMemcpyHostToDevice));
		//done later

		// Loading random number generator
		hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

		CUDA_CHECK(hipMalloc((void **)&d_Rand, np*sizeof(float)));

		printf(" ...done\n");

		printf("Create textures on GPU memory... ");
		// Copy velocity arrays
		CUDA_CHECK(hipMallocArray(&Ux_gp, &channelDescU, nx, ny));
		CUDA_CHECK(hipMallocArray(&Vx_gp, &channelDescV, nx, ny));

		CUDA_CHECK(hipMemcpyToArray(Ux_gp, 0, 0, Uo, nx*ny* sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToArray(Vx_gp, 0, 0, Vo, nx*ny* sizeof(float), hipMemcpyHostToDevice));

		texU.addressMode[0] = hipAddressModeWrap;
		texU.addressMode[1] = hipAddressModeWrap;
		texU.filterMode = hipFilterModeLinear;
		texU.normalized = false;


		CUDA_CHECK(hipBindTextureToArray(texU, Ux_gp, channelDescU));

		texV.addressMode[0] = hipAddressModeWrap;
		texV.addressMode[1] = hipAddressModeWrap;
		texV.filterMode = hipFilterModeLinear;
		texV.normalized = false;

		CUDA_CHECK(hipBindTextureToArray(texV, Vx_gp, channelDescV));

		CUDA_CHECK(hipMallocArray(&distX_gp, &channelDescdX, nx, ny));
		//CUDA_CHECK( hipMallocArray( &distXV_gp, &channelDescdXV, netav, nxiv ));
		//CUDA_CHECK( hipMallocArray( &distYU_gp, &channelDescdYU, netau, nxiu ));
		CUDA_CHECK(hipMallocArray(&distY_gp, &channelDescdY, nx, ny));

		CUDA_CHECK(hipMallocArray(&xcoord_gp, &channelDescxcoord, nx, ny));
		CUDA_CHECK(hipMallocArray(&ycoord_gp, &channelDescycoord, nx, ny));
		//CUDA_CHECK( hipMallocArray( &lon_vgp, &channelDesclonv, netav, nxiv ));
		//CUDA_CHECK( hipMallocArray( &lat_vgp, &channelDesclatv, netav, nxiv ));

		CUDA_CHECK(hipMemcpyToArray(distX_gp, 0, 0, distX, nx*ny* sizeof(float), hipMemcpyHostToDevice));
		//CUDA_CHECK( hipMemcpyToArray(distYU_gp, 0, 0, distYU, netau*nxiu* sizeof(float), hipMemcpyHostToDevice));
		//CUDA_CHECK( hipMemcpyToArray(distXV_gp, 0, 0, distXV, netav*nxiv* sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToArray(distY_gp, 0, 0, distY, nx*ny* sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToArray(xcoord_gp, 0, 0, xcoord, nx*ny* sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToArray(ycoord_gp, 0, 0, ycoord, nx*ny* sizeof(float), hipMemcpyHostToDevice));
		//CUDA_CHECK( hipMemcpyToArray(lon_vgp, 0, 0, lon_v, netav*nxiv* sizeof(float), hipMemcpyHostToDevice));
		//CUDA_CHECK( hipMemcpyToArray(lat_vgp, 0, 0, lat_v, netav*nxiv* sizeof(float), hipMemcpyHostToDevice));

		texlonu.addressMode[0] = hipAddressModeWrap;
		texlonu.addressMode[1] = hipAddressModeWrap;
		texlonu.filterMode = hipFilterModeLinear;
		texlonu.normalized = false;

		CUDA_CHECK(hipBindTextureToArray(texlonu, xcoord_gp, channelDescxcoord));

		texlatu.addressMode[0] = hipAddressModeWrap;
		texlatu.addressMode[1] = hipAddressModeWrap;
		texlatu.filterMode = hipFilterModeLinear;
		texlatu.normalized = false;

		CUDA_CHECK(hipBindTextureToArray(texlatu, ycoord_gp, channelDescycoord));

		texdXU.addressMode[0] = hipAddressModeWrap;
		texdXU.addressMode[1] = hipAddressModeWrap;
		texdXU.filterMode = hipFilterModeLinear;
		texdXU.normalized = false;

		CUDA_CHECK(hipBindTextureToArray(texdXU, distX_gp, channelDescdX));

		texdYV.addressMode[0] = hipAddressModeWrap;
		texdYV.addressMode[1] = hipAddressModeWrap;
		texdYV.filterMode = hipFilterModeLinear;
		texdYV.normalized = false;

		CUDA_CHECK(hipBindTextureToArray(texdYV, distY_gp, channelDescdY));

		printf(" ...done\n");



	}


	//read seed file //calculate seed position on the GPU if available
	
	readseedfile(seedfile, np, nx, ny, xcoord, ycoord, partpos);
	//Output seed information for sanity checks
	writexyz(np, nx, ny, xcoord, ycoord, partpos, "OutSeed_000T.xyz");
	//writexyz(xp, yp, zp, tp, xl, yl, npart, fileoutn);
	//create netcdf file
	
	creatncfile(ncoutfile, nx, ny, np, xcoord, ycoord, 0.0f, Nincel, cNincel, cTincel, partpos);


	//Run CPU/GPU loop
	totaltime = 0.0f;
	
	if (partmode > 0)
	{
		if (GPUDEV < 0) //CPU mainloop
		{
			printf("Model starting using CPU. dt=%f; \n", dt);
			printf("step %f of %f\n", totaltime, hddt*(hdend - hdstart));
			while ((hddt*hdend - totaltime) > 0.0f)
			{
				dt = min(dt, nextouttime - totaltime);
				CPUstep();
				totaltime = totaltime + dt;
				stp++;

				if ((nextouttime - totaltime) < 0.001f) // Round off error checking
				{
					//WriteoutCPU();
					char fileoutn[15];
					sprintf(fileoutn, "Part_%d.xyz", stp);
					writexyz(np, nx, ny, xcoord, ycoord, partpos, fileoutn);
					//writestep2nc(ncoutfile, nx, ny, totaltime, Nincel, cNincel, cTincel);
					writestep2nc(ncoutfile, nx, ny, np, totaltime, xcoord, ycoord, Nincel, cNincel, cTincel, partpos);
					nextouttime = nextouttime + outtime;
					dt = olddt;
					//reset Nincel 
					resetNincelCPU(nx, ny, Nincel);
				}


			}
			printf("Model Completed\n Total Number of step:%d\t total nuber of outputs steps:%d\n", stp, 0);

		}
		else //GPU main loop
		{
			//Initial particle position transfert to GPU
			printf("Copy Particle position to GPU.\n");
			CUDA_CHECK(hipMemcpy(partpos_g, partpos, np*sizeof(float4), hipMemcpyHostToDevice));

			printf("Model starting using GPU.\n");
			while ((hddt*hdend - totaltime) > 0.0f)
			{
				dt = min(dt, nextouttime - totaltime);
				//printf("dt=%f.\n",dt);
				GPUstep();
				totaltime = totaltime + dt;
				stp++;

				if ((nextouttime - totaltime) < 0.001f) // Round off error checking
				{
					//WriteoutCPU();
					char fileoutn[15];
					sprintf(fileoutn, "Part_%d.xyz", stp);
					//writexyz(np, nx, ny, xcoord, ycoord, partpos, fileoutn);
					//writestep2nc(ncoutfile, nx, ny, totaltime, Nincel, cNincel, cTincel);
					CUDA_CHECK(hipMemcpy(partpos, partpos_g, np*sizeof(float4), hipMemcpyDeviceToHost));
					CUDA_CHECK(hipMemcpy(Nincel, Nincel_g, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
					CUDA_CHECK(hipMemcpy(cNincel, cNincel_g, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
					CUDA_CHECK(hipMemcpy(cTincel, cTincel_g, nx*ny*sizeof(float), hipMemcpyDeviceToHost));


					writestep2nc(ncoutfile, nx, ny, np, totaltime, xcoord, ycoord, Nincel, cNincel, cTincel, partpos);
					nextouttime = nextouttime + outtime;
					dt = olddt;
					//reset Nincel 
					resetNincelCPU(nx, ny, Nincel);
				}


			}
		}
	}
	//Close and clean up
    
	fclose(logfile);
    return 0;
}

